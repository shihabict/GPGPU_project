#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <algorithm>

#define BLOCK_SIZE 16
#define RADIUS 1

// adaptive threshold kernel
__global__ void adaptiveThresholdKernel(const unsigned char* input, unsigned char* output, int width, int height, int blockSize, int C) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int pad = blockSize / 2;
    int sum = 0;
    int count = 0;

    for (int i = -pad; i <= pad; ++i) {
        for (int j = -pad; j <= pad; ++j) {
            int xi = x + i;
            int yj = y + j;

            if (xi >= 0 && xi < width && yj >= 0 && yj < height) {
                sum += input[yj * width + xi];
                count++;
            }
        }
    }

    int mean = sum / count;
    int threshold = mean - C;

    output[y * width + x] = (input[y * width + x] > threshold) ? 255 : 0;
}

bool readMetaFile(const std::string& meta_file_path, int& image_width, int& image_height) {
    std::ifstream meta_file(meta_file_path);
    if (!meta_file.is_open()) {
        std::cerr << "Error: Could not open metadata file!" << std::endl;
        return false;
    }

    meta_file >> image_width >> image_height;

    meta_file.close();
    return true;
}

//load raw image from file
unsigned char* loadRawImage(const std::string& raw_image_path, int image_width, int image_height) {

    std::ifstream file(raw_image_path, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open raw image file!" << std::endl;
        return nullptr;
    }

    // Memory allocation for the image
    unsigned char* image_data = new unsigned char[image_width * image_height];

    // Image to the array
    file.read(reinterpret_cast<char*>(image_data), image_width * image_height * sizeof(unsigned char));
    file.close();

    return image_data;
}

void createDirectory(const std::string& path) {
    struct stat st;
    if (stat(path.c_str(), &st) != 0) {
        mkdir(path.c_str(), 0777);
    }
}

std::string getBaseName(const std::string& path) {
    size_t last_slash = path.find_last_of("/\\");
    size_t last_dot = path.find_last_of(".");

    if (last_slash == std::string::npos) last_slash = 0;
    else last_slash++;

    if (last_dot == std::string::npos || last_dot < last_slash) {
        return path.substr(last_slash);
    }
    return path.substr(last_slash, last_dot - last_slash);
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <raw_image_path> <meta_file_path>" << std::endl;
        return -1;
    }

    std::string raw_image_path = argv[1];
    std::string meta_file_path = argv[2];

    std::string base_name = getBaseName(raw_image_path);

    createDirectory("output_images");

    createDirectory("metrices");

    int image_width = 0;
    int image_height = 0;
    if (!readMetaFile(meta_file_path, image_width, image_height)) {
        return -1;
    }

    unsigned char* h_input = loadRawImage(raw_image_path, image_width, image_height);
    if (h_input == nullptr) {
        return -1;
    }

    unsigned char* h_output = new unsigned char[image_width * image_height];

    unsigned char *d_input, *d_output;
    hipMalloc((void**)&d_input, image_width * image_height * sizeof(unsigned char));
    hipMalloc((void**)&d_output, image_width * image_height * sizeof(unsigned char));

    // Copy input image to device
    float h2d_ms = 0.0f;
    hipEvent_t h2d_start, h2d_stop;
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_stop);
    hipEventRecord(h2d_start);

    hipMemcpy(d_input, h_input, image_width * image_height * sizeof(unsigned char), hipMemcpyHostToDevice);

    hipEventRecord(h2d_stop);
    hipEventSynchronize(h2d_stop);
    hipEventElapsedTime(&h2d_ms, h2d_start, h2d_stop);
    std::cout << "Host to Device Copy Time: " << h2d_ms << " ms" << std::endl;

    // block and grid sizes
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((image_width + blockSize.x - 1) / blockSize.x, (image_height + blockSize.y - 1) / blockSize.y);

    int C = 2;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch the adaptive threshold kernel
    adaptiveThresholdKernel<<<gridSize, blockSize>>>(d_input, d_output, image_width, image_height, BLOCK_SIZE, C);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float d2h_ms = 0.0f;
    hipEvent_t d2h_start, d2h_stop;
    hipEventCreate(&d2h_start);
    hipEventCreate(&d2h_stop);
    hipEventRecord(d2h_start);

    hipMemcpy(h_output, d_output, image_width * image_height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord(d2h_stop);
    hipEventSynchronize(d2h_stop);
    hipEventElapsedTime(&d2h_ms, d2h_start, d2h_stop);
    std::cout << "Device to Host Copy Time: " << d2h_ms << " ms" << std::endl;
    std::string output_image_path = "output_images/" + base_name + "_adpt_threshold_cuda.raw";
    std::ofstream output_file(output_image_path, std::ios::binary);
    output_file.write(reinterpret_cast<char*>(h_output), image_width * image_height * sizeof(unsigned char));
    output_file.close();

    std::string metrics_path = "metrices/" + base_name + "_performance_metrics_cuda.txt";
    std::ofstream metrics_file(metrics_path);
    metrics_file << "Input Image: " << base_name << "\n";
    metrics_file << "Execution Time: " << milliseconds << " ms\n";
    metrics_file << "Host to Device Copy Time: " << h2d_ms << " ms\n";
    metrics_file << "Device to Host Copy Time: " << d2h_ms << " ms\n";

    metrics_file << "Output Image: " << output_image_path << std::endl;
    metrics_file.close();

    hipFree(d_input);
    hipFree(d_output);

    delete[] h_input;
    delete[] h_output;

    std::cout << "Adaptive Thresholding completed successfully!" << std::endl;
    std::cout << "Execution Time: " << milliseconds << " ms" << std::endl;

    return 0;
}


