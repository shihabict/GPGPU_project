#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <sys/stat.h>  // For directory creation
#include <algorithm>   // For string manipulation

#define BLOCK_SIZE 16
#define RADIUS 1 // For a 3x3 neighborhood

__global__ void adaptiveThresholdKernel(const unsigned char* input, unsigned char* output, int width, int height, int blockSize, int C) {
    // Thread and block coordinates
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    //int sharedSize = blockSize + 2 * RADIUS;
    __shared__ unsigned char tile[BLOCK_SIZE + 2 * RADIUS][BLOCK_SIZE + 2 * RADIUS];

    // Global coordinates of the shared memory element
    int sharedX = tx + RADIUS;
    int sharedY = ty + RADIUS;

    // Load the central data
    if (x < width && y < height)
        tile[sharedY][sharedX] = input[y * width + x];
    else
        tile[sharedY][sharedX] = 0;

    // Load the halo (padding) data
    if (tx < RADIUS) {
        // Left
        int x_left = x - RADIUS;
        tile[sharedY][tx] = (x_left >= 0 && y < height) ? input[y * width + x_left] : 0;
        // Right
        int x_right = x + BLOCK_SIZE;
        tile[sharedY][sharedX + RADIUS] = (x_right < width && y < height) ? input[y * width + x_right] : 0;
    }
    if (ty < RADIUS) {
        // Top
        int y_top = y - RADIUS;
        tile[ty][sharedX] = (y_top >= 0 && x < width) ? input[y_top * width + x] : 0;
        // Bottom
        int y_bottom = y + BLOCK_SIZE;
        tile[sharedY + RADIUS][sharedX] = (y_bottom < height && x < width) ? input[y_bottom * width + x] : 0;
    }

    // Corners
    if (tx < RADIUS && ty < RADIUS) {
        int x_corner = x - RADIUS;
        int y_corner = y - RADIUS;
        tile[ty][tx] = (x_corner >= 0 && y_corner >= 0) ? input[y_corner * width + x_corner] : 0;

        x_corner = x + BLOCK_SIZE;
        tile[ty][sharedX + RADIUS] = (x_corner < width && y_corner >= 0) ? input[y_corner * width + x_corner] : 0;

        y_corner = y + BLOCK_SIZE;
        tile[sharedY + RADIUS][tx] = (x_corner >= 0 && y_corner < height) ? input[y_corner * width + x_corner] : 0;

        tile[sharedY + RADIUS][sharedX + RADIUS] = (x_corner < width && y_corner < height) ? input[y_corner * width + x_corner] : 0;
    }

    __syncthreads();  // Ensure all shared memory is loaded

    // Perform local mean filtering from shared memory
    if (x < width && y < height) {
        int sum = 0;
        int count = 0;

        for (int j = -RADIUS; j <= RADIUS; ++j) {
            for (int i = -RADIUS; i <= RADIUS; ++i) {
                sum += tile[sharedY + j][sharedX + i];
                count++;
            }
        }

        int mean = sum / count;
        int threshold = mean - C;
        output[y * width + x] = (tile[sharedY][sharedX] > threshold) ? 255 : 0;
    }
}


// Function to read image dimensions from metadata file
bool readMetaFile(const std::string& meta_file_path, int& image_width, int& image_height) {
    std::ifstream meta_file(meta_file_path);
    if (!meta_file.is_open()) {
        std::cerr << "Error: Could not open metadata file!" << std::endl;
        return false;
    }

    // Read the width and height from the metadata file
    meta_file >> image_width >> image_height;

    meta_file.close();
    return true;
}

// Function to load raw image from file
unsigned char* loadRawImage(const std::string& raw_image_path, int image_width, int image_height) {
    // Open raw image file in binary mode
    std::ifstream file(raw_image_path, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open raw image file!" << std::endl;
        return nullptr;
    }

    // Allocate memory for the image
    unsigned char* image_data = new unsigned char[image_width * image_height];

    // Read image data into the array
    file.read(reinterpret_cast<char*>(image_data), image_width * image_height * sizeof(unsigned char));
    file.close();

    return image_data;
}


// Helper function to create directory if it doesn't exist
void createDirectory(const std::string& path) {
    struct stat st;
    if (stat(path.c_str(), &st) != 0) {
        mkdir(path.c_str(), 0777);
    }
}

// Helper function to extract base name from path
std::string getBaseName(const std::string& path) {
    size_t last_slash = path.find_last_of("/\\");
    size_t last_dot = path.find_last_of(".");

    if (last_slash == std::string::npos) last_slash = 0;
    else last_slash++;

    if (last_dot == std::string::npos || last_dot < last_slash) {
        return path.substr(last_slash);
    }
    return path.substr(last_slash, last_dot - last_slash);
}

int main(int argc, char* argv[]) {
    // Paths to the raw image and metadata files
//     std::string raw_image_path = "detection.raw";
//     std::string meta_file_path = "detection.raw.meta";
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <raw_image_path> <meta_file_path>" << std::endl;
        return -1;
    }

    std::string raw_image_path = argv[1];
    std::string meta_file_path = argv[2];

    // Get base name for output files
    std::string base_name = getBaseName(raw_image_path);

    // Create output directory
    createDirectory("output_images");

    // Create output directory
    createDirectory("metrices");

    // Load image dimensions from metadata file
    int image_width = 0;
    int image_height = 0;
    if (!readMetaFile(meta_file_path, image_width, image_height)) {
        return -1;
    }

    // Load raw image using the dimensions from metadata
    unsigned char* h_input = loadRawImage(raw_image_path, image_width, image_height);
    if (h_input == nullptr) {
        return -1;
    }

    // Allocate host memory for output image
    unsigned char* h_output = new unsigned char[image_width * image_height];

    // Allocate device memory
    unsigned char *d_input, *d_output;
    hipMalloc((void**)&d_input, image_width * image_height * sizeof(unsigned char));
    hipMalloc((void**)&d_output, image_width * image_height * sizeof(unsigned char));

    // Copy input image to device
   // cudaMemcpy(d_input, h_input, image_width * image_height * sizeof(unsigned char), cudaMemcpyHostToDevice);
    float h2d_ms = 0.0f;
    hipEvent_t h2d_start, h2d_stop;
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_stop);
    hipEventRecord(h2d_start);

    hipMemcpy(d_input, h_input, image_width * image_height * sizeof(unsigned char), hipMemcpyHostToDevice);

    hipEventRecord(h2d_stop);
    hipEventSynchronize(h2d_stop);
    hipEventElapsedTime(&h2d_ms, h2d_start, h2d_stop);
    std::cout << "Host to Device Copy Time: " << h2d_ms << " ms" << std::endl;

    // Define block and grid sizes
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((image_width + blockSize.x - 1) / blockSize.x, (image_height + blockSize.y - 1) / blockSize.y);

    // Threshold constant (C) and block size (to adjust the neighborhood area)
    int C = 2;
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);

    // Launch the adaptive threshold kernel
    adaptiveThresholdKernel<<<gridSize, blockSize>>>(d_input, d_output, image_width, image_height, BLOCK_SIZE, C);

    // Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy output image back to host
    //cudaMemcpy(h_output, d_output, image_width * image_height * sizeof(unsigned char), cudaMemcpyDeviceToHost);
    float d2h_ms = 0.0f;
    hipEvent_t d2h_start, d2h_stop;
    hipEventCreate(&d2h_start);
    hipEventCreate(&d2h_stop);
    hipEventRecord(d2h_start);

    hipMemcpy(h_output, d_output, image_width * image_height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord(d2h_stop);
    hipEventSynchronize(d2h_stop);
    hipEventElapsedTime(&d2h_ms, d2h_start, d2h_stop);
    std::cout << "Device to Host Copy Time: " << d2h_ms << " ms" << std::endl;

//     // Save output image (as raw file or any other format)
//     std::ofstream output_file("output_cuda_adpt_image.raw", std::ios::binary);
//     output_file.write(reinterpret_cast<char*>(h_output), image_width * image_height * sizeof(unsigned char));
//     output_file.close();
    // Save output image with base name in output_images directory
    std::string output_image_path = "output_images/" + base_name + "_adpt_threshold_cuda.raw";
    std::ofstream output_file(output_image_path, std::ios::binary);
    output_file.write(reinterpret_cast<char*>(h_output), image_width * image_height * sizeof(unsigned char));
    output_file.close();

//     // Save performance metrics to a text file
//     std::ofstream metrics_file("cuda_performance_metrics.txt");
//     metrics_file << "Execution Time: " << milliseconds << " ms" << std::endl;
//     metrics_file.close();
    // Save performance metrics with base name
    std::string metrics_path = "metrices/" + base_name + "_performance_metrics_cuda.txt";
    std::ofstream metrics_file(metrics_path);
    metrics_file << "Input Image: " << base_name << "\n";
    metrics_file << "Execution Time: " << milliseconds << " ms\n";
    metrics_file << "Host to Device Copy Time: " << h2d_ms << " ms\n";
    metrics_file << "Device to Host Copy Time: " << d2h_ms << " ms\n";

    metrics_file << "Output Image: " << output_image_path << std::endl;
    metrics_file.close();

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    delete[] h_input;
    delete[] h_output;

    std::cout << "Adaptive Thresholding completed successfully!" << std::endl;
    std::cout << "Execution Time: " << milliseconds << " ms" << std::endl;

    return 0;
}


