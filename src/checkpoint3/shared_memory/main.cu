#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <sys/stat.h> 
#include <algorithm>

#define BLOCK_SIZE 16
#define RADIUS 1

__global__ void adaptiveThresholdKernel(const unsigned char* input, unsigned char* output, int width, int height, int blockSize, int C) {
    // Thread and block coordinates
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    __shared__ unsigned char tile[BLOCK_SIZE + 2 * RADIUS][BLOCK_SIZE + 2 * RADIUS];

    int sharedX = tx + RADIUS;
    int sharedY = ty + RADIUS;

    if (x < width && y < height)
        tile[sharedY][sharedX] = input[y * width + x];
    else
        tile[sharedY][sharedX] = 0;

    if (tx < RADIUS) {
        // Left
        int x_left = x - RADIUS;
        tile[sharedY][tx] = (x_left >= 0 && y < height) ? input[y * width + x_left] : 0;
        // Right
        int x_right = x + BLOCK_SIZE;
        tile[sharedY][sharedX + RADIUS] = (x_right < width && y < height) ? input[y * width + x_right] : 0;
    }
    if (ty < RADIUS) {
        // Top
        int y_top = y - RADIUS;
        tile[ty][sharedX] = (y_top >= 0 && x < width) ? input[y_top * width + x] : 0;
        // Bottom
        int y_bottom = y + BLOCK_SIZE;
        tile[sharedY + RADIUS][sharedX] = (y_bottom < height && x < width) ? input[y_bottom * width + x] : 0;
    }

    // Corners
    if (tx < RADIUS && ty < RADIUS) {
        int x_corner = x - RADIUS;
        int y_corner = y - RADIUS;
        tile[ty][tx] = (x_corner >= 0 && y_corner >= 0) ? input[y_corner * width + x_corner] : 0;

        x_corner = x + BLOCK_SIZE;
        tile[ty][sharedX + RADIUS] = (x_corner < width && y_corner >= 0) ? input[y_corner * width + x_corner] : 0;

        y_corner = y + BLOCK_SIZE;
        tile[sharedY + RADIUS][tx] = (x_corner >= 0 && y_corner < height) ? input[y_corner * width + x_corner] : 0;

        tile[sharedY + RADIUS][sharedX + RADIUS] = (x_corner < width && y_corner < height) ? input[y_corner * width + x_corner] : 0;
    }

    __syncthreads(); 

    if (x < width && y < height) {
        int sum = 0;
        int count = 0;

        for (int j = -RADIUS; j <= RADIUS; ++j) {
            for (int i = -RADIUS; i <= RADIUS; ++i) {
                sum += tile[sharedY + j][sharedX + i];
                count++;
            }
        }

        int mean = sum / count;
        int threshold = mean - C;
        output[y * width + x] = (tile[sharedY][sharedX] > threshold) ? 255 : 0;
    }
}


bool readMetaFile(const std::string& meta_file_path, int& image_width, int& image_height) {
    std::ifstream meta_file(meta_file_path);
    if (!meta_file.is_open()) {
        std::cerr << "Error: Could not open metadata file!" << std::endl;
        return false;
    }

    meta_file >> image_width >> image_height;

    meta_file.close();
    return true;
}

unsigned char* loadRawImage(const std::string& raw_image_path, int image_width, int image_height) {

    std::ifstream file(raw_image_path, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open raw image file!" << std::endl;
        return nullptr;
    }

    unsigned char* image_data = new unsigned char[image_width * image_height];

    file.read(reinterpret_cast<char*>(image_data), image_width * image_height * sizeof(unsigned char));
    file.close();

    return image_data;
}



void createDirectory(const std::string& path) {
    struct stat st;
    if (stat(path.c_str(), &st) != 0) {
        mkdir(path.c_str(), 0777);
    }
}


std::string getBaseName(const std::string& path) {
    size_t last_slash = path.find_last_of("/\\");
    size_t last_dot = path.find_last_of(".");

    if (last_slash == std::string::npos) last_slash = 0;
    else last_slash++;

    if (last_dot == std::string::npos || last_dot < last_slash) {
        return path.substr(last_slash);
    }
    return path.substr(last_slash, last_dot - last_slash);
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <raw_image_path> <meta_file_path>" << std::endl;
        return -1;
    }

    std::string raw_image_path = argv[1];
    std::string meta_file_path = argv[2];

    std::string base_name = getBaseName(raw_image_path);

    createDirectory("output_images");

    createDirectory("metrices");

    int image_width = 0;
    int image_height = 0;
    if (!readMetaFile(meta_file_path, image_width, image_height)) {
        return -1;
    }

    unsigned char* h_input = loadRawImage(raw_image_path, image_width, image_height);
    if (h_input == nullptr) {
        return -1;
    }

    unsigned char* h_output = new unsigned char[image_width * image_height];

    unsigned char *d_input, *d_output;
    hipMalloc((void**)&d_input, image_width * image_height * sizeof(unsigned char));
    hipMalloc((void**)&d_output, image_width * image_height * sizeof(unsigned char));

    float h2d_ms = 0.0f;
    hipEvent_t h2d_start, h2d_stop;
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_stop);
    hipEventRecord(h2d_start);

    hipMemcpy(d_input, h_input, image_width * image_height * sizeof(unsigned char), hipMemcpyHostToDevice);

    hipEventRecord(h2d_stop);
    hipEventSynchronize(h2d_stop);
    hipEventElapsedTime(&h2d_ms, h2d_start, h2d_stop);
    std::cout << "Host to Device Copy Time: " << h2d_ms << " ms" << std::endl;

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((image_width + blockSize.x - 1) / blockSize.x, (image_height + blockSize.y - 1) / blockSize.y);

    int C = 2;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    adaptiveThresholdKernel<<<gridSize, blockSize>>>(d_input, d_output, image_width, image_height, BLOCK_SIZE, C);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);


    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float d2h_ms = 0.0f;
    hipEvent_t d2h_start, d2h_stop;
    hipEventCreate(&d2h_start);
    hipEventCreate(&d2h_stop);
    hipEventRecord(d2h_start);

    hipMemcpy(h_output, d_output, image_width * image_height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord(d2h_stop);
    hipEventSynchronize(d2h_stop);
    hipEventElapsedTime(&d2h_ms, d2h_start, d2h_stop);
    std::cout << "Device to Host Copy Time: " << d2h_ms << " ms" << std::endl;

    std::string output_image_path = "output_images/" + base_name + "_adpt_threshold_cuda.raw";
    std::ofstream output_file(output_image_path, std::ios::binary);
    output_file.write(reinterpret_cast<char*>(h_output), image_width * image_height * sizeof(unsigned char));
    output_file.close();

    std::string metrics_path = "metrices/" + base_name + "_performance_metrics_cuda.txt";
    std::ofstream metrics_file(metrics_path);
    metrics_file << "Input Image: " << base_name << "\n";
    metrics_file << "Execution Time: " << milliseconds << " ms\n";
    metrics_file << "Host to Device Copy Time: " << h2d_ms << " ms\n";
    metrics_file << "Device to Host Copy Time: " << d2h_ms << " ms\n";

    metrics_file << "Output Image: " << output_image_path << std::endl;
    metrics_file.close();

    hipFree(d_input);
    hipFree(d_output);

    delete[] h_input;
    delete[] h_output;

    std::cout << "Adaptive Thresholding completed successfully!" << std::endl;
    std::cout << "Execution Time: " << milliseconds << " ms" << std::endl;

    return 0;
}


