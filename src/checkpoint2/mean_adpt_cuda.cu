#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <algorithm> 

#define BLOCK_SIZE 16
#define RADIUS 1 // For a 3x3 neighborhood

// Adaptive threshold kernel
__global__ void adaptiveThresholdKernel(const unsigned char* input, unsigned char* output, int width, int height, int blockSize, int C) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int pad = blockSize / 2;
    int sum = 0;
    int count = 0;

    // Loop through the blockSize area to calculate the local neighborhood sum
    for (int i = -pad; i <= pad; ++i) {
        for (int j = -pad; j <= pad; ++j) {
            int xi = x + i;
            int yj = y + j;

            if (xi >= 0 && xi < width && yj >= 0 && yj < height) {
                sum += input[yj * width + xi];
                count++;
            }
        }
    }

    int mean = sum / count;
    int threshold = mean - C;

    output[y * width + x] = (input[y * width + x] > threshold) ? 255 : 0;
}

// Function to read image dimensions from metadata
bool readMetaFile(const std::string& meta_file_path, int& image_width, int& image_height) {
    std::ifstream meta_file(meta_file_path);
    if (!meta_file.is_open()) {
        std::cerr << "Error: Could not open metadata file!" << std::endl;
        return false;
    }

    // Read the width and height
    meta_file >> image_width >> image_height;

    meta_file.close();
    return true;
}

// load raw image from file
unsigned char* loadRawImage(const std::string& raw_image_path, int image_width, int image_height) {
    // Open raw image
    std::ifstream file(raw_image_path, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open raw image file!" << std::endl;
        return nullptr;
    }

    // Allocate memory for the image
    unsigned char* image_data = new unsigned char[image_width * image_height];

    // image data into the array
    file.read(reinterpret_cast<char*>(image_data), image_width * image_height * sizeof(unsigned char));
    file.close();

    return image_data;
}


// function to create directory
void createDirectory(const std::string& path) {
    struct stat st;
    if (stat(path.c_str(), &st) != 0) {
        mkdir(path.c_str(), 0777);
    }
}

//function to extract base name from path
std::string getBaseName(const std::string& path) {
    size_t last_slash = path.find_last_of("/\\");
    size_t last_dot = path.find_last_of(".");

    if (last_slash == std::string::npos) last_slash = 0;
    else last_slash++;

    if (last_dot == std::string::npos || last_dot < last_slash) {
        return path.substr(last_slash);
    }
    return path.substr(last_slash, last_dot - last_slash);
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "usage: " << argv[0] << " <raw_image_path> <meta_file_path>" << std::endl;
        return -1;
    }

    std::string raw_image_path = argv[1];
    std::string meta_file_path = argv[2];

    // base name for output files
    std::string base_name = getBaseName(raw_image_path);

    // Create output directory
    createDirectory("output_images");

    // Create metrics directory
    createDirectory("metrices");

    // Load image dimensions from metadata file
    int image_width = 0;
    int image_height = 0;
    if (!readMetaFile(meta_file_path, image_width, image_height)) {
        return -1;
    }

    // Load raw image using the dimensions from metadata
    unsigned char* h_input = loadRawImage(raw_image_path, image_width, image_height);
    if (h_input == nullptr) {
        return -1;
    }

    // host memory allocation for the output image
    unsigned char* h_output = new unsigned char[image_width * image_height];

    // device memory allocation
    unsigned char *d_input, *d_output;
    hipMalloc((void**)&d_input, image_width * image_height * sizeof(unsigned char));
    hipMalloc((void**)&d_output, image_width * image_height * sizeof(unsigned char));

    // input image to device
    hipMemcpy(d_input, h_input, image_width * image_height * sizeof(unsigned char), hipMemcpyHostToDevice);

    //  block and grid sizes
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((image_width + blockSize.x - 1) / blockSize.x, (image_height + blockSize.y - 1) / blockSize.y);

    // Threshold constant c 
    int C = 2;
    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start timing
    hipEventRecord(start);

    // launch adaptive threshold kernel 
    adaptiveThresholdKernel<<<gridSize, blockSize>>>(d_input, d_output, image_width, image_height, BLOCK_SIZE, C);

    // kernel errors checking
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //  elapsed time calculation
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // output image back to host
    hipMemcpy(h_output, d_output, image_width * image_height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    std::string output_image_path = "output_images/" + base_name + "_adpt_threshold_cuda.raw";
    std::ofstream output_file(output_image_path, std::ios::binary);
    output_file.write(reinterpret_cast<char*>(h_output), image_width * image_height * sizeof(unsigned char));
    output_file.close();

    // performance metrics with base name
    std::string metrics_path = "metrices/" + base_name + "_performance_metrics_cuda.txt";
    std::ofstream metrics_file(metrics_path);
    metrics_file << "Input Image: " << base_name << "\n";
    metrics_file << "Execution Time: " << milliseconds << " ms\n";
    metrics_file << "Output Image: " << output_image_path << std::endl;
    metrics_file.close();

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    delete[] h_input;
    delete[] h_output;

    std::cout << "Adaptive Thresholding completed successfully!" << std::endl;
    std::cout << "Execution Time: " << milliseconds << " ms" << std::endl;

    return 0;
}


